#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

__global__ void kernel_double(int *c, int *a)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = a[i] * 2;
}
int main()
{
    const int size = 100;
    int a[size][size], c[size][size];
	int sum_a = 0;
	int sum_c = 0;

	for (int i = 0; i < size; i++) {
		for (int j = 0; j < size; j++) {
			a[i][j] = rand() % 10;
			sum_a += a[i][j];
		}
	}
	printf("sum of matrix a is %d \n", sum_a);

	int *dev_a = 0;
	int *dev_c = 0;
	hipMalloc((void**)&dev_c, size * size * sizeof(int));
	hipMalloc((void**)&dev_a, size * size * sizeof(int));
	hipMemcpy(dev_a, a, size * size * sizeof(int), hipMemcpyHostToDevice);
	printf("grid size %d \n", int(size * size / 1024) + 1);
	kernel_double << <int(size * size / 1024) + 1, 1024  >> >(dev_c, dev_a);
	hipDeviceSynchronize();
	hipMemcpy(c, dev_c, size * size * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_c);
	hipFree(dev_a);
	for (int i = 0; i < size; i++) {
		for (int j = 0; j < size; j++) {
			sum_c += c[i][j];
		}
	}
	printf("sum of matrix c is %d \n", sum_c);
	return 0;
}
